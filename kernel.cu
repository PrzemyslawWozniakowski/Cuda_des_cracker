#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdlib>
#include <iostream>
#include <chrono>
#include <random>
#include <ctime>
#include <cstdio>
#include "Constants.cuh"
#include "deviceAndHostFunctions.cuh"
#include "deviceFunctions.cuh"
#include "hostFunctions.cuh"

#pragma region CUDA_WRAPPERS

void cudaCheckErrors(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess) {
		std::cout << hipGetErrorString(cudaStatus) << std::endl;
		exit(1);
	}
}
#pragma endregion

__host__ void Crack_Host(uint64_t* crackedKey, uint64_t dataToEncrypt, uint64_t encryptedMessage, uint64_t maxKeyVal, int keyLength);
__host__ void PrintUint(uint64_t v);
__global__ void Crack_Kernel(uint64_t data, uint64_t encodedData, uint64_t *crackedkey, bool *foundFlag, uint64_t maxKeyVal, int keyLength);
__host__ uint64_t GenerateDesKey(int keyLenght);

int main()
{

	cudaCheckErrors(hipSetDevice(0));
	std::cout << "Dlugosc klucza:" << std::endl;

	int keyLength;
	std::cin >> keyLength;

	uint64_t maxKeyVal;

	if (keyLength == 64)
		maxKeyVal = UINT64_MAX;
	else
	{
		maxKeyVal = (uint64_t)1 << keyLength;
		maxKeyVal -= 1;
	}
	uint64_t desKey = GenerateDesKey(keyLength);
	uint64_t dataToEncrypt = 0x0123456789ABCDEF;
	uint64_t encryptedMessage = EncryptData(dataToEncrypt, desKey);

	std::cout << "Wiadomosc: ";
	PrintUint(dataToEncrypt);
	std::cout << "Wiadomosc po zaszyfrowaniu: ";
	PrintUint(encryptedMessage);

	uint64_t* deviceKey = NULL, crackedKeyGPU;
	int cracked_val = 0;
	bool *wasCracked = NULL;

	cudaCheckErrors(hipMalloc((void**)&deviceKey, sizeof(uint64_t)));
	cudaCheckErrors(hipMalloc((void**)&wasCracked, sizeof(int)));
	cudaCheckErrors(hipMemcpy(wasCracked, &cracked_val, sizeof(int), hipMemcpyHostToDevice));

	std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
	Crack_Kernel << <4096, 1024 >> > (dataToEncrypt, encryptedMessage, deviceKey, wasCracked, maxKeyVal, keyLength);
	cudaCheckErrors(hipDeviceSynchronize());
	std::chrono::system_clock::time_point end = std::chrono::system_clock::now();

	auto gpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000000.0;

	cudaCheckErrors(hipMemcpy(&crackedKeyGPU, deviceKey, sizeof(uint64_t), hipMemcpyDeviceToHost));

	uint64_t encryptedDataWithKeyFromGPU = EncryptData(dataToEncrypt, crackedKeyGPU);
	if (encryptedDataWithKeyFromGPU == encryptedMessage)
	{
		std::cout << "GPU klucz znaleziony w: " << gpuExecutionTime << " sekund" << std::endl;
		std::cout << "Klucz znaleziony na GPU: " << (crackedKeyGPU >> (MAXL - keyLength));
		PrintUint(crackedKeyGPU);
		std::cout << "Oryginalny klucz: " << (desKey >> (MAXL - keyLength));
		PrintUint(desKey);
		std::cout << "Wiadomosc po zaszyfrowaniu kluczem z GPU: ";
		PrintUint(encryptedDataWithKeyFromGPU);
	}
	else if (crackedKeyGPU == 0)
	{
		std::cout << "GPU nie znalazlo klucza." << std::endl << std::endl;
	}
	else
	{
		std::cout << "GPU klucz nie dziala." << std::endl;
	}
	std::cout << "================================================= " << std::endl << std::endl;

	start = std::chrono::system_clock::now();
	uint64_t crackedKeyCPU = -1;
	Crack_Host(&crackedKeyCPU, dataToEncrypt, encryptedMessage, maxKeyVal, keyLength);

	end = std::chrono::system_clock::now();

	auto cpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000000.0;

	if (crackedKeyCPU != -1)
	{
		std::cout << "CPU klucz znaleziony w: " << cpuExecutionTime << " sekund" << std::endl;
		std::cout << "Klucz znaleziony na CPU: " << (crackedKeyCPU >> (MAXL - keyLength));
		PrintUint(crackedKeyCPU);
		std::cout << "Oryginalny klucz: " << (desKey >> (MAXL - keyLength));
		PrintUint(desKey);
	}
	else
	{
		std::cout << "CPU klucz nie dziala." << std::endl;
	}

	std::cout << "GPU znajduje klucz w " << gpuExecutionTime / cpuExecutionTime * 100 << " % czasu CPU." << std::endl;

	hipFree(deviceKey);
	hipFree(wasCracked);

	return 0;
}

__global__ void Crack_Kernel(uint64_t data, uint64_t encodedData, uint64_t *crackedkey, bool *foundFlag, uint64_t maxKeyVal, int keyLength)
{
	for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i <= maxKeyVal; i += blockDim.x * gridDim.x)
	{
		uint64_t keycandidate = i << (MAXL - keyLength);
		uint64_t currentValue = EncryptData_Device(data, keycandidate);
		if (currentValue == encodedData)
		{
			*crackedkey = keycandidate;
			*foundFlag = false;
			return;
		}
		if (*foundFlag == true)
		{
			return;
		}
	}
}

__host__ void Crack_Host(uint64_t* crackedKey, uint64_t dataToEncrypt, uint64_t encryptedMessage, uint64_t maxKeyVal, int keyLength)
{
	for (uint64_t i = 0; i <= maxKeyVal; i++)
	{
		uint64_t keycandidate = i << (MAXL - keyLength);
		uint64_t currentValue = EncryptData(dataToEncrypt, keycandidate);
		if (currentValue == encryptedMessage)
		{
			*crackedKey = keycandidate;
			break;
		}
	}
}

__host__ void PrintUint(uint64_t v)
{
	std::cout << "\n";
	uint64_t j = 1;
	for (int i = 0; i < 64; i++)
	{
		std::cout << (v >> (63 - i) &j);
		if ((i + 1) % 8 == 0)
			std::cout << " ";
	}
	std::cout << "\n";
}


#pragma endregion