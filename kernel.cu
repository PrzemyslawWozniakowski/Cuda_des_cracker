
#include "hip/hip_runtime.h"


#include <cstdlib>
#include <iostream>
#include <chrono>
#include <random>
//#include "Const_Values.h"

typedef unsigned long long uint64;
#define MAXL 64
#pragma region CUDA_CONSTANTS

__constant__ int PC_1[56] = {
	57,	49,	41,	33,	25,	17,	9,
	1,	58,	50,	42,	34,	26,	18,
	10,	2,	59,	51,	43,	35,	27,
	19,	11,	3,	60,	52,	44,	36,
	63,	55,	47,	39,	31,	23,	15,
	7,	62,	54,	46,	38,	30,	22,
	14,	6,	61,	53,	45,	37,	29,
	21,	13,	5,	28,	20,	12,	4
};

__constant__ int PC_2[48] = {
	14, 17, 11, 24, 1,	5,
	3,	28, 15, 6,	21, 10,
	23, 19, 12, 4,	26, 8,
	16, 7,	27, 20, 13, 2,
	41, 52, 31, 37, 47, 55,
	30, 40, 51, 45, 33, 48,
	44, 49, 39, 56, 34, 53,
	46, 42, 50, 36, 29, 32
};

__constant__ int IP[64] = {
	58,	50,	42,	34,	26,	18,	10,	2,
	60,	52,	44,	36,	28,	20,	12,	4,
	62,	54,	46,	38,	30,	22,	14,	6,
	64,	56,	48,	40,	32,	24,	16,	8,
	57,	49,	41,	33,	25,	17,	 9,	1,
	59,	51,	43,	35,	27,	19,	11,	3,
	61,	53,	45,	37,	29,	21,	13,	5,
	63,	55,	47,	39,	31,	23,	15,	7
};

__constant__ int E_BIT[48] = {
	32,	1,	2,	3,	4,	5,
	4,	5,	6,	7,	8,	9,
	8,	9,	10,	11,	12,	13,
	12,	13,	14,	15,	16,	17,
	16,	17,	18,	19,	20,	21,
	20,	21,	22,	23,	24,	25,
	24,	25,	26,	27,	28,	29,
	28,	29,	30,	31,	32,	1
};

__constant__ int S1[64] = {
	14,	4,	13,	1,	2,	15,	11,	8,	3,	10,	6,	12,	5,	9,	0,	7,
	0,	15,	7,	4,	14,	2,	13,	1,	10,	6,	12,	11,	9,	5,	3,	8,
	4,	1,	14,	8,	13,	6,	2,	11,	15,	12,	9,	7,	3,	10,	5,	0,
	15,	12,	8,	2,	4,	9,	1,	7,	5,	11,	3,	14,	10,	0,	6,	13
};

__constant__ int S2[64] = {
	15,	1,	8,	14,	6,	11,	3,	4,	9,	7,	2,	13,	12,	0,	5,	10,
	3,	13,	4,	7,	15,	2,	8,	14,	12,	0,	1,	10,	6,	9,	11,	5,
	0,	14,	7,	11,	10,	4,	13,	1,	5,	8,	12,	6,	9,	3,	2,	15,
	13,	8,	10,	1,	3,	15,	4,	2,	11,	6,	7,	12,	0,	5,	14,	9,
};

__constant__ int S3[64] = {
	10,	0,	9,	14,	6,	3,	15,	5,	1,	13,	12,	7,	11,	4,	2,	8,
	13,	7,	0,	9,	3,	4,	6,	10,	2,	8,	5,	14,	12,	11,	15,	1,
	13,	6,	4,	9,	8,	15,	3,	0,	11,	1,	2,	12,	5,	10,	14,	7,
	1,	10,	13,	0,	6,	9,	8,	7,	4,	15,	14,	3,	11,	5,	2,	12
};

__constant__ int S4[64] = {
	7,	13,	14,	3,	0,	6,	9,	10,	1,	2,	8,	5,	11,	12,	4,	15,
	13,	8,	11,	5,	6,	15,	0,	3,	4,	7,	2,	12,	1,	10,	14,	9,
	10,	6,	9,	0,	12,	11,	7,	13,	15,	1,	3,	14,	5,	2,	8,	4,
	3,	15,	0,	6,	10,	1,	13,	8,	9,	4,	5,	11,	12,	7,	2,	14
};

__constant__ int S5[64] = {
	2,	12,	4,	1,	7,	10,	11,	6,	8,	5,	3,	15,	13,	0,	14,	9,
	14,	11,	2,	12,	4,	7,	13,	1,	5,	0,	15,	10,	3,	9,	8,	6,
	4,	2,	1,	11,	10,	13,	7,	8,	15,	9,	12,	5,	6,	3,	0,	14,
	11,	8,	12,	7,	1,	14,	2,	13,	6,	15,	0,	9,	10,	4,	5,	3
};

__constant__ int S6[64] = {
	12,	1,	10,	15,	9,	2,	6,	8,	0,	13,	3,	4,	14,	7,	5,	11,
	10,	15,	4,	2,	7,	12,	9,	5,	6,	1,	13,	14,	0,	11,	3,	8,
	9,	14,	15,	5,	2,	8,	12,	3,	7,	0,	4,	10,	1,	13,	11,	6,
	4,	3,	2,	12,	9,	5,	15,	10,	11,	14,	1,	7,	6,	0,	8,	13,
};

__constant__ int S7[64] = {
	4,	11,	2,	14,	15,	0,	8,	13,	3,	12,	9,	7,	5,	10,	6,	1,
	13,	0,	11,	7,	4,	9,	1,	10,	14,	3,	5,	12,	2,	15,	8,	6,
	1,	4,	11,	13,	12,	3,	7,	14,	10,	15,	6,	8,	0,	5,	9,	2,
	6,	11,	13,	8,	1,	4,	10,	7,	9,	5,	0,	15,	14,	2,	3,	12,
};

__constant__ int S8[64] = {
	13,	2,	8,	4,	6,	15,	11,	1,	10,	9,	3,	14,	5,	0,	12,	7,
	1,	15,	13,	8,	10,	3,	7,	4,	12,	5,	6,	11,	0,	14,	9,	2,
	7,	11,	4,	1,	9,	12,	14,	2,	0,	6,	10,	13,	15,	3,	5,	8,
	2,	1,	14,	7,	4,	10,	8,	13,	15,	12,	9,	0,	3,	5,	6,	11,
};

__constant__ int* ALL_S[8] = {
	S1, S2, S3, S4, S5, S6, S7, S8
};

__constant__ int P[32] = {
	16,	7,	20, 21,
	29,	12, 28, 17,
	1,	15, 23, 26,
	5,	18, 31, 10,
	2,	8,	24, 14,
	32, 27, 3,	9,
	19, 13, 30,	6,
	22, 11, 4,	25
};

__constant__ int IP_REV[64] = {
	40,	8, 48, 16, 56, 24, 64, 32,
	39, 7, 47, 15, 55, 23, 63, 31,
	38, 6, 46, 14, 54, 22, 62, 30,
	37, 5, 45, 13, 53, 21, 61, 29,
	36, 4, 44, 12, 52, 20, 60, 28,
	35, 3, 43, 11, 51, 19, 59, 27,
	34, 2, 42, 10, 50, 18, 58, 26,
	33, 1, 41,	9, 49, 17, 57, 25
};

__constant__ int SHIFTS[16] = {
	1,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1
};

#pragma endregion

#pragma region HOST_CONSTANTS

int PC_1_HOST[56] = {
	57,	49,	41,	33,	25,	17,	9,
	1,	58,	50,	42,	34,	26,	18,
	10,	2,	59,	51,	43,	35,	27,
	19,	11,	3,	60,	52,	44,	36,
	63,	55,	47,	39,	31,	23,	15,
	7,	62,	54,	46,	38,	30,	22,
	14,	6,	61,	53,	45,	37,	29,
	21,	13,	5,	28,	20,	12,	4
};

int PC_2_HOST[48] = {
	14, 17, 11, 24, 1,	5,
	3,	28, 15, 6,	21, 10,
	23, 19, 12, 4,	26, 8,
	16, 7,	27, 20, 13, 2,
	41, 52, 31, 37, 47, 55,
	30, 40, 51, 45, 33, 48,
	44, 49, 39, 56, 34, 53,
	46, 42, 50, 36, 29, 32
};

int IP_HOST[64] = {
	58,	50,	42,	34,	26,	18,	10,	2,
	60,	52,	44,	36,	28,	20,	12,	4,
	62,	54,	46,	38,	30,	22,	14,	6,
	64,	56,	48,	40,	32,	24,	16,	8,
	57,	49,	41,	33,	25,	17,	 9,	1,
	59,	51,	43,	35,	27,	19,	11,	3,
	61,	53,	45,	37,	29,	21,	13,	5,
	63,	55,	47,	39,	31,	23,	15,	7
};

int E_BIT_HOST[48] = {
	32,	1,	2,	3,	4,	5,
	4,	5,	6,	7,	8,	9,
	8,	9,	10,	11,	12,	13,
	12,	13,	14,	15,	16,	17,
	16,	17,	18,	19,	20,	21,
	20,	21,	22,	23,	24,	25,
	24,	25,	26,	27,	28,	29,
	28,	29,	30,	31,	32,	1
};

int S1_HOST[64] = {
	14,	4,	13,	1,	2,	15,	11,	8,	3,	10,	6,	12,	5,	9,	0,	7,
	0,	15,	7,	4,	14,	2,	13,	1,	10,	6,	12,	11,	9,	5,	3,	8,
	4,	1,	14,	8,	13,	6,	2,	11,	15,	12,	9,	7,	3,	10,	5,	0,
	15,	12,	8,	2,	4,	9,	1,	7,	5,	11,	3,	14,	10,	0,	6,	13
};

int S2_HOST[64] = {
	15,	1,	8,	14,	6,	11,	3,	4,	9,	7,	2,	13,	12,	0,	5,	10,
	3,	13,	4,	7,	15,	2,	8,	14,	12,	0,	1,	10,	6,	9,	11,	5,
	0,	14,	7,	11,	10,	4,	13,	1,	5,	8,	12,	6,	9,	3,	2,	15,
	13,	8,	10,	1,	3,	15,	4,	2,	11,	6,	7,	12,	0,	5,	14,	9,
};

int S3_HOST[64] = {
	10,	0,	9,	14,	6,	3,	15,	5,	1,	13,	12,	7,	11,	4,	2,	8,
	13,	7,	0,	9,	3,	4,	6,	10,	2,	8,	5,	14,	12,	11,	15,	1,
	13,	6,	4,	9,	8,	15,	3,	0,	11,	1,	2,	12,	5,	10,	14,	7,
	1,	10,	13,	0,	6,	9,	8,	7,	4,	15,	14,	3,	11,	5,	2,	12
};

int S4_HOST[64] = {
	7,	13,	14,	3,	0,	6,	9,	10,	1,	2,	8,	5,	11,	12,	4,	15,
	13,	8,	11,	5,	6,	15,	0,	3,	4,	7,	2,	12,	1,	10,	14,	9,
	10,	6,	9,	0,	12,	11,	7,	13,	15,	1,	3,	14,	5,	2,	8,	4,
	3,	15,	0,	6,	10,	1,	13,	8,	9,	4,	5,	11,	12,	7,	2,	14
};

int S5_HOST[64] = {
	2,	12,	4,	1,	7,	10,	11,	6,	8,	5,	3,	15,	13,	0,	14,	9,
	14,	11,	2,	12,	4,	7,	13,	1,	5,	0,	15,	10,	3,	9,	8,	6,
	4,	2,	1,	11,	10,	13,	7,	8,	15,	9,	12,	5,	6,	3,	0,	14,
	11,	8,	12,	7,	1,	14,	2,	13,	6,	15,	0,	9,	10,	4,	5,	3
};

int S6_HOST[64] = {
	12,	1,	10,	15,	9,	2,	6,	8,	0,	13,	3,	4,	14,	7,	5,	11,
	10,	15,	4,	2,	7,	12,	9,	5,	6,	1,	13,	14,	0,	11,	3,	8,
	9,	14,	15,	5,	2,	8,	12,	3,	7,	0,	4,	10,	1,	13,	11,	6,
	4,	3,	2,	12,	9,	5,	15,	10,	11,	14,	1,	7,	6,	0,	8,	13,
};

int S7_HOST[64] = {
	4,	11,	2,	14,	15,	0,	8,	13,	3,	12,	9,	7,	5,	10,	6,	1,
	13,	0,	11,	7,	4,	9,	1,	10,	14,	3,	5,	12,	2,	15,	8,	6,
	1,	4,	11,	13,	12,	3,	7,	14,	10,	15,	6,	8,	0,	5,	9,	2,
	6,	11,	13,	8,	1,	4,	10,	7,	9,	5,	0,	15,	14,	2,	3,	12,
};

int S8_HOST[64] = {
	13,	2,	8,	4,	6,	15,	11,	1,	10,	9,	3,	14,	5,	0,	12,	7,
	1,	15,	13,	8,	10,	3,	7,	4,	12,	5,	6,	11,	0,	14,	9,	2,
	7,	11,	4,	1,	9,	12,	14,	2,	0,	6,	10,	13,	15,	3,	5,	8,
	2,	1,	14,	7,	4,	10,	8,	13,	15,	12,	9,	0,	3,	5,	6,	11,
};

int* ALL_S_HOST[8] = {
	S1_HOST, S2_HOST, S3_HOST, S4_HOST, S5_HOST, S6_HOST, S7_HOST, S8_HOST
};

int P_HOST[32] = {
	16,	7,	20, 21,
	29,	12, 28, 17,
	1,	15, 23, 26,
	5,	18, 31, 10,
	2,	8,	24, 14,
	32, 27, 3,	9,
	19, 13, 30,	6,
	22, 11, 4,	25
};

int IP_REV_HOST[64] = {
	40,	8, 48, 16, 56, 24, 64, 32,
	39, 7, 47, 15, 55, 23, 63, 31,
	38, 6, 46, 14, 54, 22, 62, 30,
	37, 5, 45, 13, 53, 21, 61, 29,
	36, 4, 44, 12, 52, 20, 60, 28,
	35, 3, 43, 11, 51, 19, 59, 27,
	34, 2, 42, 10, 50, 18, 58, 26,
	33, 1, 41,	9, 49, 17, 57, 25
};

int SHIFTS_HOST[16] = {
	1,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1
};
#pragma endregion

#pragma region CUDA_WRAPPERS

void cudaCheckErrors(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess) {
		std::cout << hipGetErrorString(cudaStatus) << std::endl;
		exit(1);
	}
}
#pragma endregion

__device__ __host__ uint64 Get_Bit(uint64 number, int bitNumber);
__device__ __host__ void Set_Bit(uint64* number, int bitNumber, uint64 value);
__device__ __host__ uint64 Permute(uint64 number, int* Permutation_Table, int length);
__device__ __host__ void Split(uint64 key, uint64* left, uint64* right, int keyLength);
__device__ __host__ uint64 CycleBitsToLeft(uint64 value, int shiftNumber, int valueLength);

__host__ uint64 EncryptData_Host(uint64 dataToEncrypt, uint64 desKey);
__host__  void Create_SubKeys_Host(uint64* subKeys, uint64 desKey);
__host__  void Create_Kn_Host(uint64* subkeys, uint64* C, uint64* D);
__host__ uint64 Function_Host(uint64 data, uint64 key);
__host__ uint64 Encode_Host(uint64* subKeys, uint64 dataToEncrypt);


__device__ uint64 EncryptData(uint64 dataToEncrypt, uint64 desKey);
__device__  void Create_SubKeys(uint64* subKeys, uint64 desKey);
__device__  uint64* Create_Kn(uint64* C, uint64* D);
__device__ uint64 Function(uint64 data, uint64 key);
__device__ uint64 Encode(uint64* subKeys, uint64 dataToEncrypt);

__host__ void PrintUint(uint64 v);

__global__ void Crack(uint64 data, uint64 encodedData, uint64 *key, bool *done, uint64 maxLenght);
__host__ uint64 GenerateDesKey(int keyLenght);


int main()
{
	uint64 dataToEncrypt = 0x0123456789ABCDEF;
	uint64 desKey = 0x133457799BBCDFF1;

	uint64 encryptedData = EncryptData_Host(dataToEncrypt, desKey);
	/*cudaCheckErrors(cudaSetDevice(0));
	std::cout << "Enter key lenght:" << std::endl;

	int keyLenght;
	std::cin >> keyLenght;
	int maxLenght = 1 << keyLenght;

	uint64 desKey = GenerateDesKey(keyLenght);
	uint64 dataToEncrypt = 0x0123456789ABCDEF;
	uint64 encryptedData = EncryptData_Host(dataToEncrypt, desKey);

	uint64* devKey = NULL, crackedKey;
	int done_val = 0;
	bool *done = NULL;
	cudaCheckErrors(cudaMalloc((void**)&devKey, sizeof(uint64)));
	cudaCheckErrors(cudaMalloc((void**)&done, sizeof(int)));
	cudaCheckErrors(cudaMemcpy(done, &done_val, sizeof(int), cudaMemcpyHostToDevice));

	std::chrono::system_clock::time_point begin = std::chrono::system_clock::now();
	Crack << <16, 16 >> > (dataToEncrypt, encryptedData, devKey, done, maxLenght);

	cudaCheckErrors(cudaDeviceSynchronize());
	std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
	auto gpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) / 1000000.0;

	cudaCheckErrors(cudaMemcpy(&crackedKey, devKey, sizeof(uint64), cudaMemcpyDeviceToHost));
	uint64 encryptedDataWithKeyFromGPU = EncryptData_Host(dataToEncrypt, crackedKey);
	if (encryptedDataWithKeyFromGPU == encryptedData)
	{
		std::cout << "[GPU] Found matching key in " << gpuExecutionTime << " seconds" << std::endl;
		std::cout << "Found key: " << crackedKey << std::endl;
		std::cout << "Original key: " << desKey << std::endl << std::endl;
	}
	else if (crackedKey == 0)
	{
		std::cout << "[GPU] Can not find matching key!" << std::endl << std::endl;
	}
	else
	{
		std::cout << "[GPU] Found key do not work!" << std::endl;
	}


	begin = std::chrono::system_clock::now();
	int keyFound = -1;
	for (uint64 i = 0; i < maxLenght; i++)
	{
		uint64 currentValue = EncryptData_Host(dataToEncrypt, i);
		if (currentValue == encryptedData)
		{
			keyFound = i;
			break;
		}
	}
	end = std::chrono::system_clock::now();
	auto cpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) / 1000000.0;

	if (keyFound != -1)
	{
		std::cout << "[CPU] Found matching key in " << cpuExecutionTime << " sekund" << std::endl;
		std::cout << "Found key: " << keyFound << std::endl;
		std::cout << "Original key: " << desKey << std::endl;
	}
	else
	{
		std::cout << "[CPU] Can not find matching key!" << std::endl;
	}

	std::cout << "GPU solving time is " << gpuExecutionTime / cpuExecutionTime * 100 << " % CPU solving time." << std::endl;

	cudaFree(devKey);
	cudaFree(done);*/

	return 0;
}

__global__ void Crack(uint64 data, uint64 encodedData, uint64 *key, bool *foundKey, uint64 maxLenght)
{
	for (uint64 i = blockIdx.x * blockDim.x + threadIdx.x; i <= maxLenght; i += blockDim.x * gridDim.x)
	{
		uint64 currentValue = EncryptData(data, i);
		if (currentValue == encodedData)
		{
			*key = i;
			*foundKey = false;
			return;
		}
		if (*foundKey == true)
		{
			return;
		}
	}
}

#pragma region DeviceAndHostFunctions

__host__ void PrintUint(uint64 v)
{
	std::cout << "\n";
	uint64 j = 1;
	for (int i = 0; i < 64; i++)
	{
		std::cout << (v>>(63-i) &j);
	}
	std::cout << "\n";

}

__device__ __host__ uint64 Get_Bit(uint64 number, int bitNumber)
{
	return ((uint64)1 & (number>>(MAXL-bitNumber)));
}

__device__ __host__ void Set_Bit(uint64* number, int bitNumber, uint64 value)
{
	(*number) = (*number) &  ~((uint64)1 << (MAXL - bitNumber));
	(*number)= (*number) | (value << (MAXL - bitNumber));
}

__device__ __host__ uint64 Permute(uint64 number, int* Permutation_Table, int length)
{
	uint64 numberchanged = 0;
	for (int i = 0; i < length; i++)
	{
		Set_Bit(&numberchanged, i+1, Get_Bit(number, Permutation_Table[i]));
	}
	return numberchanged;	
}

__device__ __host__ void Split(uint64 key, uint64* left, uint64* right, int keyLength)
{
	*right = *left = 0;
	for (int i = 1; i <= keyLength / 2; i++)
	{
		Set_Bit(right, i, Get_Bit(key, keyLength / 2 + i));
		Set_Bit(left, i, Get_Bit(key, i));
	}
}

__device__ __host__ uint64 CycleBitsToLeft(uint64 value, int shiftNumber, int valueLength) 
{
	for (int i = 0; i < shiftNumber; i++)
	{
		uint64 bit = Get_Bit(value, 1);
		value <<= 1;
		Set_Bit(&value, valueLength, bit);
	}
	return value;
}

#pragma endregion

#pragma region DeviceFunctions

__device__ uint64 EncryptData(uint64 dataToEncrypt, uint64 desKey)
{
	uint64 subKeys[16];
	Create_SubKeys(subKeys, desKey);
	return Encode(subKeys, dataToEncrypt);
}

__device__  void Create_SubKeys(uint64* subKeys, uint64 desKey)
{
	uint64 kplus = Permute(desKey, PC_1, 56);

	uint64 C[17];
	uint64 D[17];

	Split(kplus, &C[0], &D[0], 56);

	for (int i = 1; i <= 16; i++)
	{
		C[i] = CycleBitsToLeft(C[i - 1], SHIFTS[i], 28);
		D[i] = CycleBitsToLeft(D[i - 1], SHIFTS[i], 28);
	}

	subKeys= Create_Kn(C, D);
}

__device__  uint64* Create_Kn(uint64* C, uint64* D)
{
	uint64	Kn[16];
	for(int i=0;i<16;i++)
	{
		Kn[i] = C[i + 1];
		Kn[i] = Kn[i] | (D[i + 1]>>28);
		Kn[i] = Permute(Kn[i], PC_2, 48);
	}
	return Kn;
}

__device__ uint64 Encode(uint64* subKeys,uint64 data)
{
	uint64 data_ip = Permute(data, IP, 64);

	uint64 L[17];
	uint64 R[17];

	Split(data_ip, &L[0], &R[0], 64);

	for (int i = 1; i <= 16; i++)
	{
		
		L[i] = R[i-1];
		R[i] = L[i - 1] ^ Function(R[i - 1], subKeys[i - 1]);
	}
	uint64 RL = R[16] | (L[16]>>32);

	return Permute(RL, IP_REV, 64);
}

__device__ uint64 Function(uint64 data, uint64 key)
{
	uint64 ER = Permute(data, E_BIT, 48);
	uint64 KxorER = ER ^ key;
	uint64 S[8];
	uint64 B[8];

	for (int i = 0; i < 8; i++)
	{
		B[i] = 0;

		for (int j = 1; j <= 6; j++)
		{
			Set_Bit(&B[i], j, Get_Bit(KxorER, i * 6 + j));
		}

		uint64 firstAndLastBit = Get_Bit(B[i], 6) << 1 | Get_Bit(B[i], 1);
		uint64 middleBits = Get_Bit(B[i], 5) << 3 | Get_Bit(B[i], 5) << 2 | Get_Bit(B[i], 3) << 1 | Get_Bit(B[i], 2);

		S[i] = ALL_S[i][(int)firstAndLastBit * 16 + (int)middleBits];
	}
	uint64 result = 0;

	for (int i = 0; i < 8; i++)
	{
		result |= S[i] << 28 - 4 * i;
	}

	return Permute(result, P, 32);
}

#pragma endregion

#pragma region HostFunctions

///
__host__ uint64 GenerateDesKey(int keyLenght)
{
	std::mt19937 mt;
	std::uniform_int_distribution<int> randomV(0, 1);

	uint64 key = 0;
	for (int i = 0; i < keyLenght; i++)
	{
		Set_Bit(&key, MAXL-i, randomV(mt));
	}
	return key;
}
///
__host__  uint64 EncryptData_Host(uint64 dataToEncrypt, uint64 desKey)
{
	uint64 subKeys[16];

	Create_SubKeys_Host(subKeys, desKey);
	PrintUint(subKeys[0]);
	return Encode_Host(subKeys, dataToEncrypt);
}

__host__  void Create_SubKeys_Host(uint64* subKeys, uint64 desKey)
{
	uint64 kplus = Permute(desKey, PC_1_HOST, 56);
	//PrintUint(kplus);
	uint64 C[17];
	uint64 D[17];

	Split(kplus, &C[0], &D[0], 56);

	for (int i = 1; i <= 16; i++)
	{
		C[i] = CycleBitsToLeft(C[i - 1], SHIFTS_HOST[i-1], 28);
		D[i] = CycleBitsToLeft(D[i - 1], SHIFTS_HOST[i-1], 28);
	}
	//PrintUint(C[0]);
	//PrintUint(D[0]);

	//PrintUint(C[1]);
	//PrintUint(D[1]);
	//PrintUint(C[2]);
	//PrintUint(D[2]);
	//PrintUint(C[3]);
	//PrintUint(D[3]);
	//PrintUint(C[16]);
	//PrintUint(D[16]);
	
	Create_Kn_Host(subKeys,C, D);
	//PrintUint(subKeys[0]);
	//PrintUint(D[16]);
}

__host__  void Create_Kn_Host(uint64* subkeys, uint64* C, uint64* D)
{
	
	for (int i = 0; i < 16; i++)
	{
		subkeys[i] = C[i + 1];
		subkeys[i] = subkeys[i] | (D[i + 1] >> 28);
		subkeys[i] = Permute(subkeys[i], PC_2_HOST, 48);
	}
	
}

__host__  uint64 Encode_Host(uint64* subKeys, uint64 data)
{
	uint64 data_ip = Permute(data, IP_HOST, 64);

	uint64 L[17];
	uint64 R[17];

	Split(data_ip, &L[0], &R[0], 64);
	//PrintUint(L[0]);
	//PrintUint(R[0]);


	for (int i = 1; i <= 16; i++)
	{

		L[i] = R[i - 1];
		R[i] = L[i - 1] ^ Function_Host(R[i - 1], subKeys[i - 1]);

	}
	uint64 RL = R[16] | (L[16] >> 32);

	return Permute(RL, IP_REV_HOST, 64);
}

__host__  uint64 Function_Host(uint64 data, uint64 key)
{
	uint64 ER = Permute(data, E_BIT_HOST, 48);
	uint64 KxorER = ER ^ key;
	uint64 S[8];
	uint64 B[8];

	PrintUint(KxorER);
	PrintUint(KxorER);
	//// FINISHED JERE
	for (int i = 0; i < 8; i++)
	{
		B[i] = 0;

		for (int j = 1; j <= 6; j++)
		{
			Set_Bit(&B[i], j, Get_Bit(KxorER, i * 6 + j));
		}

		uint64 firstAndLastBit = Get_Bit(B[i], 6) << 1 | Get_Bit(B[i], 1);
		uint64 middleBits = Get_Bit(B[i], 5) << 3 | Get_Bit(B[i], 5) << 2 | Get_Bit(B[i], 3) << 1 | Get_Bit(B[i], 2);

		S[i] = ALL_S_HOST[i][(int)firstAndLastBit * 16 + (int)middleBits];
	}
	uint64 result = 0;

	for (int i = 0; i < 8; i++)
	{
		result |= S[i] << 28 - 4 * i;
	}

	return Permute(result, P_HOST, 32);
}

#pragma endregion