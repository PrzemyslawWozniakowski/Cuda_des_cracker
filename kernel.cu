#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdlib>
#include <iostream>
#include <chrono>
#include <random>
#include <ctime>
#include <cstdio>


#define MAXL 64
#pragma region CUDA_CONSTANTS

__constant__ int PC_1[56] = {
	57,	49,	41,	33,	25,	17,	9,
	1,	58,	50,	42,	34,	26,	18,
	10,	2,	59,	51,	43,	35,	27,
	19,	11,	3,	60,	52,	44,	36,
	63,	55,	47,	39,	31,	23,	15,
	7,	62,	54,	46,	38,	30,	22,
	14,	6,	61,	53,	45,	37,	29,
	21,	13,	5,	28,	20,	12,	4
};

__constant__ int PC_2[48] = {
	14, 17, 11, 24, 1,	5,
	3,	28, 15, 6,	21, 10,
	23, 19, 12, 4,	26, 8,
	16, 7,	27, 20, 13, 2,
	41, 52, 31, 37, 47, 55,
	30, 40, 51, 45, 33, 48,
	44, 49, 39, 56, 34, 53,
	46, 42, 50, 36, 29, 32
};

__constant__ int IP[64] = {
	58,	50,	42,	34,	26,	18,	10,	2,
	60,	52,	44,	36,	28,	20,	12,	4,
	62,	54,	46,	38,	30,	22,	14,	6,
	64,	56,	48,	40,	32,	24,	16,	8,
	57,	49,	41,	33,	25,	17,	 9,	1,
	59,	51,	43,	35,	27,	19,	11,	3,
	61,	53,	45,	37,	29,	21,	13,	5,
	63,	55,	47,	39,	31,	23,	15,	7
};

__constant__ int E_BIT[48] = {
	32,	1,	2,	3,	4,	5,
	4,	5,	6,	7,	8,	9,
	8,	9,	10,	11,	12,	13,
	12,	13,	14,	15,	16,	17,
	16,	17,	18,	19,	20,	21,
	20,	21,	22,	23,	24,	25,
	24,	25,	26,	27,	28,	29,
	28,	29,	30,	31,	32,	1
};

__constant__ int S1[64] = {
	14,	4,	13,	1,	2,	15,	11,	8,	3,	10,	6,	12,	5,	9,	0,	7,
	0,	15,	7,	4,	14,	2,	13,	1,	10,	6,	12,	11,	9,	5,	3,	8,
	4,	1,	14,	8,	13,	6,	2,	11,	15,	12,	9,	7,	3,	10,	5,	0,
	15,	12,	8,	2,	4,	9,	1,	7,	5,	11,	3,	14,	10,	0,	6,	13
};

__constant__ int S2[64] = {
	15,	1,	8,	14,	6,	11,	3,	4,	9,	7,	2,	13,	12,	0,	5,	10,
	3,	13,	4,	7,	15,	2,	8,	14,	12,	0,	1,	10,	6,	9,	11,	5,
	0,	14,	7,	11,	10,	4,	13,	1,	5,	8,	12,	6,	9,	3,	2,	15,
	13,	8,	10,	1,	3,	15,	4,	2,	11,	6,	7,	12,	0,	5,	14,	9,
};

__constant__ int S3[64] = {
	10,	0,	9,	14,	6,	3,	15,	5,	1,	13,	12,	7,	11,	4,	2,	8,
	13,	7,	0,	9,	3,	4,	6,	10,	2,	8,	5,	14,	12,	11,	15,	1,
	13,	6,	4,	9,	8,	15,	3,	0,	11,	1,	2,	12,	5,	10,	14,	7,
	1,	10,	13,	0,	6,	9,	8,	7,	4,	15,	14,	3,	11,	5,	2,	12
};

__constant__ int S4[64] = {
	7,	13,	14,	3,	0,	6,	9,	10,	1,	2,	8,	5,	11,	12,	4,	15,
	13,	8,	11,	5,	6,	15,	0,	3,	4,	7,	2,	12,	1,	10,	14,	9,
	10,	6,	9,	0,	12,	11,	7,	13,	15,	1,	3,	14,	5,	2,	8,	4,
	3,	15,	0,	6,	10,	1,	13,	8,	9,	4,	5,	11,	12,	7,	2,	14
};

__constant__ int S5[64] = {
	2,	12,	4,	1,	7,	10,	11,	6,	8,	5,	3,	15,	13,	0,	14,	9,
	14,	11,	2,	12,	4,	7,	13,	1,	5,	0,	15,	10,	3,	9,	8,	6,
	4,	2,	1,	11,	10,	13,	7,	8,	15,	9,	12,	5,	6,	3,	0,	14,
	11,	8,	12,	7,	1,	14,	2,	13,	6,	15,	0,	9,	10,	4,	5,	3
};

__constant__ int S6[64] = {
	12,	1,	10,	15,	9,	2,	6,	8,	0,	13,	3,	4,	14,	7,	5,	11,
	10,	15,	4,	2,	7,	12,	9,	5,	6,	1,	13,	14,	0,	11,	3,	8,
	9,	14,	15,	5,	2,	8,	12,	3,	7,	0,	4,	10,	1,	13,	11,	6,
	4,	3,	2,	12,	9,	5,	15,	10,	11,	14,	1,	7,	6,	0,	8,	13,
};

__constant__ int S7[64] = {
	4,	11,	2,	14,	15,	0,	8,	13,	3,	12,	9,	7,	5,	10,	6,	1,
	13,	0,	11,	7,	4,	9,	1,	10,	14,	3,	5,	12,	2,	15,	8,	6,
	1,	4,	11,	13,	12,	3,	7,	14,	10,	15,	6,	8,	0,	5,	9,	2,
	6,	11,	13,	8,	1,	4,	10,	7,	9,	5,	0,	15,	14,	2,	3,	12,
};

__constant__ int S8[64] = {
	13,	2,	8,	4,	6,	15,	11,	1,	10,	9,	3,	14,	5,	0,	12,	7,
	1,	15,	13,	8,	10,	3,	7,	4,	12,	5,	6,	11,	0,	14,	9,	2,
	7,	11,	4,	1,	9,	12,	14,	2,	0,	6,	10,	13,	15,	3,	5,	8,
	2,	1,	14,	7,	4,	10,	8,	13,	15,	12,	9,	0,	3,	5,	6,	11,
};

__constant__ int* ALL_S[8] = {
	S1, S2, S3, S4, S5, S6, S7, S8
};

__constant__ int P[32] = {
	16,	7,	20, 21,
	29,	12, 28, 17,
	1,	15, 23, 26,
	5,	18, 31, 10,
	2,	8,	24, 14,
	32, 27, 3,	9,
	19, 13, 30,	6,
	22, 11, 4,	25
};

__constant__ int IP_REV[64] = {
	40,	8, 48, 16, 56, 24, 64, 32,
	39, 7, 47, 15, 55, 23, 63, 31,
	38, 6, 46, 14, 54, 22, 62, 30,
	37, 5, 45, 13, 53, 21, 61, 29,
	36, 4, 44, 12, 52, 20, 60, 28,
	35, 3, 43, 11, 51, 19, 59, 27,
	34, 2, 42, 10, 50, 18, 58, 26,
	33, 1, 41,	9, 49, 17, 57, 25
};

__constant__ int SHIFTS[16] = {
	1,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1
};

#pragma endregion

#pragma region HOST_CONSTANTS

int PC_1_HOST[56] = {
	57,	49,	41,	33,	25,	17,	9,
	1,	58,	50,	42,	34,	26,	18,
	10,	2,	59,	51,	43,	35,	27,
	19,	11,	3,	60,	52,	44,	36,
	63,	55,	47,	39,	31,	23,	15,
	7,	62,	54,	46,	38,	30,	22,
	14,	6,	61,	53,	45,	37,	29,
	21,	13,	5,	28,	20,	12,	4
};

int PC_2_HOST[48] = {
	14, 17, 11, 24, 1,	5,
	3,	28, 15, 6,	21, 10,
	23, 19, 12, 4,	26, 8,
	16, 7,	27, 20, 13, 2,
	41, 52, 31, 37, 47, 55,
	30, 40, 51, 45, 33, 48,
	44, 49, 39, 56, 34, 53,
	46, 42, 50, 36, 29, 32
};

int IP_HOST[64] = {
	58,	50,	42,	34,	26,	18,	10,	2,
	60,	52,	44,	36,	28,	20,	12,	4,
	62,	54,	46,	38,	30,	22,	14,	6,
	64,	56,	48,	40,	32,	24,	16,	8,
	57,	49,	41,	33,	25,	17,	 9,	1,
	59,	51,	43,	35,	27,	19,	11,	3,
	61,	53,	45,	37,	29,	21,	13,	5,
	63,	55,	47,	39,	31,	23,	15,	7
};

int E_BIT_HOST[48] = {
	32,	1,	2,	3,	4,	5,
	4,	5,	6,	7,	8,	9,
	8,	9,	10,	11,	12,	13,
	12,	13,	14,	15,	16,	17,
	16,	17,	18,	19,	20,	21,
	20,	21,	22,	23,	24,	25,
	24,	25,	26,	27,	28,	29,
	28,	29,	30,	31,	32,	1
};

int S1_HOST[64] = {
	14,	4,	13,	1,	2,	15,	11,	8,	3,	10,	6,	12,	5,	9,	0,	7,
	0,	15,	7,	4,	14,	2,	13,	1,	10,	6,	12,	11,	9,	5,	3,	8,
	4,	1,	14,	8,	13,	6,	2,	11,	15,	12,	9,	7,	3,	10,	5,	0,
	15,	12,	8,	2,	4,	9,	1,	7,	5,	11,	3,	14,	10,	0,	6,	13
};

int S2_HOST[64] = {
	15,	1,	8,	14,	6,	11,	3,	4,	9,	7,	2,	13,	12,	0,	5,	10,
	3,	13,	4,	7,	15,	2,	8,	14,	12,	0,	1,	10,	6,	9,	11,	5,
	0,	14,	7,	11,	10,	4,	13,	1,	5,	8,	12,	6,	9,	3,	2,	15,
	13,	8,	10,	1,	3,	15,	4,	2,	11,	6,	7,	12,	0,	5,	14,	9,
};

int S3_HOST[64] = {
	10,	0,	9,	14,	6,	3,	15,	5,	1,	13,	12,	7,	11,	4,	2,	8,
	13,	7,	0,	9,	3,	4,	6,	10,	2,	8,	5,	14,	12,	11,	15,	1,
	13,	6,	4,	9,	8,	15,	3,	0,	11,	1,	2,	12,	5,	10,	14,	7,
	1,	10,	13,	0,	6,	9,	8,	7,	4,	15,	14,	3,	11,	5,	2,	12
};

int S4_HOST[64] = {
	7,	13,	14,	3,	0,	6,	9,	10,	1,	2,	8,	5,	11,	12,	4,	15,
	13,	8,	11,	5,	6,	15,	0,	3,	4,	7,	2,	12,	1,	10,	14,	9,
	10,	6,	9,	0,	12,	11,	7,	13,	15,	1,	3,	14,	5,	2,	8,	4,
	3,	15,	0,	6,	10,	1,	13,	8,	9,	4,	5,	11,	12,	7,	2,	14
};

int S5_HOST[64] = {
	2,	12,	4,	1,	7,	10,	11,	6,	8,	5,	3,	15,	13,	0,	14,	9,
	14,	11,	2,	12,	4,	7,	13,	1,	5,	0,	15,	10,	3,	9,	8,	6,
	4,	2,	1,	11,	10,	13,	7,	8,	15,	9,	12,	5,	6,	3,	0,	14,
	11,	8,	12,	7,	1,	14,	2,	13,	6,	15,	0,	9,	10,	4,	5,	3
};

int S6_HOST[64] = {
	12,	1,	10,	15,	9,	2,	6,	8,	0,	13,	3,	4,	14,	7,	5,	11,
	10,	15,	4,	2,	7,	12,	9,	5,	6,	1,	13,	14,	0,	11,	3,	8,
	9,	14,	15,	5,	2,	8,	12,	3,	7,	0,	4,	10,	1,	13,	11,	6,
	4,	3,	2,	12,	9,	5,	15,	10,	11,	14,	1,	7,	6,	0,	8,	13,
};

int S7_HOST[64] = {
	4,	11,	2,	14,	15,	0,	8,	13,	3,	12,	9,	7,	5,	10,	6,	1,
	13,	0,	11,	7,	4,	9,	1,	10,	14,	3,	5,	12,	2,	15,	8,	6,
	1,	4,	11,	13,	12,	3,	7,	14,	10,	15,	6,	8,	0,	5,	9,	2,
	6,	11,	13,	8,	1,	4,	10,	7,	9,	5,	0,	15,	14,	2,	3,	12,
};

int S8_HOST[64] = {
	13,	2,	8,	4,	6,	15,	11,	1,	10,	9,	3,	14,	5,	0,	12,	7,
	1,	15,	13,	8,	10,	3,	7,	4,	12,	5,	6,	11,	0,	14,	9,	2,
	7,	11,	4,	1,	9,	12,	14,	2,	0,	6,	10,	13,	15,	3,	5,	8,
	2,	1,	14,	7,	4,	10,	8,	13,	15,	12,	9,	0,	3,	5,	6,	11,
};

int* ALL_S_HOST[8] = {
	S1_HOST, S2_HOST, S3_HOST, S4_HOST, S5_HOST, S6_HOST, S7_HOST, S8_HOST
};

int P_HOST[32] = {
	16,	7,	20, 21,
	29,	12, 28, 17,
	1,	15, 23, 26,
	5,	18, 31, 10,
	2,	8,	24, 14,
	32, 27, 3,	9,
	19, 13, 30,	6,
	22, 11, 4,	25
};

int IP_REV_HOST[64] = {
	40,	8, 48, 16, 56, 24, 64, 32,
	39, 7, 47, 15, 55, 23, 63, 31,
	38, 6, 46, 14, 54, 22, 62, 30,
	37, 5, 45, 13, 53, 21, 61, 29,
	36, 4, 44, 12, 52, 20, 60, 28,
	35, 3, 43, 11, 51, 19, 59, 27,
	34, 2, 42, 10, 50, 18, 58, 26,
	33, 1, 41,	9, 49, 17, 57, 25
};

int SHIFTS_HOST[16] = {
	1,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1,
	2,
	2,
	2,
	2,
	2,
	2,
	1
};
#pragma endregion

#pragma region CUDA_WRAPPERS

void cudaCheckErrors(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess) {
		std::cout << hipGetErrorString(cudaStatus) << std::endl;
		exit(1);
	}
}
#pragma endregion

__device__ __host__ uint64_t GetNBit(uint64_t number, int bitNumber);
__device__ __host__ void SetNBit(uint64_t* number, int bitNumber, uint64_t value);
__device__ __host__ uint64_t ApplyPermutation(uint64_t number, int* Permutation_Table, int length);
__device__ __host__ void SplitInHalf(uint64_t key, uint64_t* left, uint64_t* right, int keyLength);
__device__ __host__ uint64_t CycleToLeft(uint64_t value, int shiftNumber, int valueLength);

__host__ uint64_t EncryptData(uint64_t dataToEncrypt, uint64_t desKey);
__host__  void GenerateSubKeys(uint64_t* subKeys, uint64_t desKey);
__host__  void GenerateKn(uint64_t* subkeys, uint64_t* C, uint64_t* D);
__host__ uint64_t Function(uint64_t data, uint64_t key);
__host__ uint64_t Encode(uint64_t* subKeys, uint64_t dataToEncrypt);
__host__ void Crack_Host(uint64_t* crackedKey, uint64_t dataToEncrypt, uint64_t encryptedMessage, uint64_t maxKeyVal, int keyLength);

__device__ uint64_t EncryptData_Device(uint64_t dataToEncrypt, uint64_t desKey);
__device__  void GenerateSubKeys_Device(uint64_t* subKeys, uint64_t desKey);
__device__  void GenerateKn_Device(uint64_t* subkeys, uint64_t* C, uint64_t* D);
__device__ uint64_t Function_Device(uint64_t data, uint64_t key);
__device__ uint64_t Encode_Device(uint64_t* subKeys, uint64_t dataToEncrypt);

__host__ void PrintUint(uint64_t v);
__global__ void Crack_Kernel(uint64_t data, uint64_t encodedData, uint64_t *crackedkey, bool *foundFlag, uint64_t maxKeyVal, int keyLength);
__host__ uint64_t GenerateDesKey(int keyLenght);


int main()
{

	cudaCheckErrors(hipSetDevice(0));
	std::cout << "Dlugosc klucza:" << std::endl;

	int keyLength;
	std::cin >> keyLength;
	uint64_t maxKeyVal = (uint64_t)1 << keyLength;
	uint64_t desKey = GenerateDesKey(keyLength);
	uint64_t dataToEncrypt = 0x0123456789ABCDEF;
	uint64_t encryptedMessage = EncryptData(dataToEncrypt, desKey);
	
	uint64_t* deviceKey = NULL, crackedKeyGPU;
	int cracked_val = 0;
	bool *wasCracked = NULL;
	cudaCheckErrors(hipMalloc((void**)&deviceKey, sizeof(uint64_t)));
	cudaCheckErrors(hipMalloc((void**)&wasCracked, sizeof(int)));
	cudaCheckErrors(hipMemcpy(wasCracked, &cracked_val, sizeof(int), hipMemcpyHostToDevice));

	std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
	Crack_Kernel << <4096, 1024 >> > (dataToEncrypt, encryptedMessage, deviceKey, wasCracked, maxKeyVal, keyLength);
	cudaCheckErrors(hipDeviceSynchronize());
	std::chrono::system_clock::time_point end = std::chrono::system_clock::now();

	auto gpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000000.0;

	cudaCheckErrors(hipMemcpy(&crackedKeyGPU, deviceKey, sizeof(uint64_t), hipMemcpyDeviceToHost));

	uint64_t encryptedDataWithKeyFromGPU = EncryptData(dataToEncrypt, crackedKeyGPU);
	if (encryptedDataWithKeyFromGPU == encryptedMessage)
	{
		std::cout << "GPU klucz znaleziony w: " << gpuExecutionTime << " sekund" << std::endl;
		std::cout << "Klucz znaleziony na GPU: " << (crackedKeyGPU >> (MAXL - keyLength));
		PrintUint(crackedKeyGPU);
		std::cout << "Oryginalny klucz: " <<(desKey >> (MAXL - keyLength));
		PrintUint(desKey);
	}
	else if (crackedKeyGPU == 0)
	{
		std::cout << "GPU nie znalazlo klucza." << std::endl << std::endl;
	}
	else
	{
		std::cout << "GPU klucz nie dziala." << std::endl;
	}
	std::cout << "================================================= " << std::endl << std::endl;


	start = std::chrono::system_clock::now();
	uint64_t crackedKeyCPU = -1;
	Crack_Host(&crackedKeyCPU, dataToEncrypt, encryptedMessage, maxKeyVal, keyLength);

	end = std::chrono::system_clock::now();

	auto cpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000000.0;

	if (crackedKeyCPU != -1)
	{
		std::cout << "CPU klucz znaleziony w: " << cpuExecutionTime << " sekund" << std::endl;
		std::cout << "Klucz znaleziony na CPU: " << (crackedKeyCPU >> (MAXL - keyLength));
		PrintUint(crackedKeyCPU);
		std::cout << "Oryginalny klucz: " << (desKey >> (MAXL - keyLength));
		PrintUint(desKey);
	}
	else
	{
		std::cout << "CPU klucz nie dziala." << std::endl;
	}

	std::cout << "GPU znajduje klucz w " << gpuExecutionTime / cpuExecutionTime * 100 << " % czasu CPU." << std::endl;

	hipFree(deviceKey);
	hipFree(wasCracked);

	return 0;
}

__global__ void Crack_Kernel(uint64_t data, uint64_t encodedData, uint64_t *crackedkey, bool *foundFlag, uint64_t maxKeyVal, int keyLength)
{
	for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i <= maxKeyVal; i += blockDim.x * gridDim.x)
	{
		uint64_t keycandidate = i << (MAXL - keyLength);
		uint64_t currentValue = EncryptData_Device(data, keycandidate);
		if (currentValue == encodedData)
		{
			*crackedkey = keycandidate;
			*foundFlag = false;
			return;
		}
		if (*foundFlag == true)
		{
			return;
		}
	}
}

__host__ void Crack_Host(uint64_t* crackedKey, uint64_t dataToEncrypt, uint64_t encryptedMessage, uint64_t maxKeyVal, int keyLength)
{
	for (uint64_t i = 0; i < maxKeyVal; i++)
	{
		uint64_t keycandidate = i << (MAXL - keyLength);
		uint64_t currentValue = EncryptData(dataToEncrypt, keycandidate);
		if (currentValue == encryptedMessage)
		{
			*crackedKey = keycandidate;
			break;
		}
	}
}

#pragma region DeviceAndHostFunctions

__host__ void PrintUint(uint64_t v)
{
	std::cout << "\n";
	uint64_t j = 1;
	for (int i = 0; i < 64; i++)
	{
		std::cout << (v>>(63-i) &j);
		if ((i + 1) % 8 == 0)
			std::cout << " ";
	}
	std::cout << "\n";

}

__device__ __host__ uint64_t GetNBit(uint64_t number, int bitNumber)
{
	return ((uint64_t)1 & (number>>(MAXL-bitNumber)));
}

__device__ __host__ void SetNBit(uint64_t* number, int bitNumber, uint64_t value)
{
	(*number) = (*number) &  ~((uint64_t)1 << (MAXL - bitNumber));
	(*number)= (*number) | (value << (MAXL - bitNumber));
}

__device__ __host__ uint64_t ApplyPermutation(uint64_t number, int* Permutation_Table, int length)
{
	uint64_t numberchanged = 0;
	for (int i = 0; i < length; i++)
	{
		SetNBit(&numberchanged, i+1, GetNBit(number, Permutation_Table[i]));
	}
	return numberchanged;	
}

__device__ __host__ void SplitInHalf(uint64_t key, uint64_t* left, uint64_t* right, int keyLength)
{
	*right = *left = 0;
	for (int i = 1; i <= keyLength / 2; i++)
	{
		SetNBit(right, i, GetNBit(key, keyLength / 2 + i));
		SetNBit(left, i, GetNBit(key, i));
	}
}

__device__ __host__ uint64_t CycleToLeft(uint64_t value, int shiftNumber, int valueLength) 
{
	for (int i = 0; i < shiftNumber; i++)
	{
		uint64_t bit = GetNBit(value, 1);
		value <<= 1;
		SetNBit(&value, valueLength, bit);
	}
	return value;
}

#pragma endregion

#pragma region DeviceFunctions


__device__  uint64_t EncryptData_Device(uint64_t dataToEncrypt, uint64_t desKey)
{
	uint64_t subKeys[16];

	GenerateSubKeys_Device(subKeys, desKey);
	uint64_t  encoded = Encode_Device(subKeys, dataToEncrypt);
	return encoded;
}

__device__  void GenerateSubKeys_Device(uint64_t* subKeys, uint64_t desKey)
{
	uint64_t kplus = ApplyPermutation(desKey, PC_1, 56);
	uint64_t C[17];
	uint64_t D[17];

	SplitInHalf(kplus, &C[0], &D[0], 56);

	for (int i = 1; i <= 16; i++)
	{
		C[i] = CycleToLeft(C[i - 1], SHIFTS[i - 1], 28);
		D[i] = CycleToLeft(D[i - 1], SHIFTS[i - 1], 28);
	}

	GenerateKn_Device(subKeys, C, D);
}

__device__  void GenerateKn_Device(uint64_t* subkeys, uint64_t* C, uint64_t* D)
{
	for (int i = 0; i < 16; i++)
	{
		subkeys[i] = C[i + 1];
		subkeys[i] = subkeys[i] | (D[i + 1] >> 28);
		subkeys[i] = ApplyPermutation(subkeys[i], PC_2, 48);
	}

}

__device__  uint64_t Encode_Device(uint64_t* subKeys, uint64_t data)
{
	uint64_t data_ip = ApplyPermutation(data, IP, 64);

	uint64_t L[17];
	uint64_t R[17];

	SplitInHalf(data_ip, &L[0], &R[0], 64);


	for (int i = 1; i <= 16; i++)
	{

		L[i] = R[i - 1];
		R[i] = L[i - 1] ^ Function_Device(R[i - 1], subKeys[i - 1]);
	}
	uint64_t RL = R[16] | (L[16] >> 32);
	return ApplyPermutation(RL, IP_REV, 64);
}

__device__  uint64_t Function_Device(uint64_t data, uint64_t key)
{
	uint64_t ER = ApplyPermutation(data, E_BIT, 48);
	uint64_t KxorER = ER ^ key;
	uint64_t S[8];
	uint64_t B[8];
	for (int i = 0; i < 8; i++)
	{
		B[i] = 0;

		for (int j = 1; j <= 6; j++)
		{
			SetNBit(&B[i], j, GetNBit(KxorER, i * 6 + j));
		}
		uint64_t firstLastBit = GetNBit(B[i], 1) << 1 | GetNBit(B[i], 6);
		uint64_t midBits = GetNBit(B[i], 2) << 3 | GetNBit(B[i], 3) << 2 | GetNBit(B[i], 4) << 1 | GetNBit(B[i], 5);
		S[i] = ALL_S[i][(int)firstLastBit * 16 + (int)midBits];
	}
	uint64_t result = 0;

	for (int i = 0; i < 8; i++)
	{
		result |= S[i] << 60 - 4 * i;

	}
	return ApplyPermutation(result, P, 32);
}


#pragma endregion

#pragma region HostFunctions

__host__ uint64_t GenerateDesKey(int keyLenght)
{
	std::mt19937 mt(time(0));
	std::uniform_int_distribution<int> randomV(0, 1);

	uint64_t key = 0;
	for (int i = 1; i <= keyLenght; i++)
	{
		SetNBit(&key,i, randomV(mt));
	}
	return key;
}

__host__  uint64_t EncryptData(uint64_t dataToEncrypt, uint64_t desKey)
{
	uint64_t subKeys[16];

	GenerateSubKeys(subKeys, desKey);
	return Encode(subKeys, dataToEncrypt);
}

__host__  void GenerateSubKeys(uint64_t* subKeys, uint64_t desKey)
{
	uint64_t kplus = ApplyPermutation(desKey, PC_1_HOST, 56);
	uint64_t C[17];
	uint64_t D[17];

	SplitInHalf(kplus, &C[0], &D[0], 56);

	for (int i = 1; i <= 16; i++)
	{
		C[i] = CycleToLeft(C[i - 1], SHIFTS_HOST[i-1], 28);
		D[i] = CycleToLeft(D[i - 1], SHIFTS_HOST[i-1], 28);
	}
	
	GenerateKn(subKeys,C, D);
}

__host__  void GenerateKn(uint64_t* subkeys, uint64_t* C, uint64_t* D)
{
	
	for (int i = 0; i < 16; i++)
	{
		subkeys[i] = C[i + 1];
		subkeys[i] = subkeys[i] | (D[i + 1] >> 28);
		subkeys[i] = ApplyPermutation(subkeys[i], PC_2_HOST, 48);
	}
	
}

__host__  uint64_t Encode(uint64_t* subKeys, uint64_t data)
{
	uint64_t data_ip = ApplyPermutation(data, IP_HOST, 64);

	uint64_t L[17];
	uint64_t R[17];

	SplitInHalf(data_ip, &L[0], &R[0], 64);


	for (int i = 1; i <= 16; i++)
	{

		L[i] = R[i - 1];
		R[i] = L[i - 1] ^ Function(R[i - 1], subKeys[i - 1]);
	}
	uint64_t RL = R[16] | (L[16] >> 32);
	return ApplyPermutation(RL, IP_REV_HOST, 64);
}

__host__  uint64_t Function(uint64_t data, uint64_t key)
{
	uint64_t ER = ApplyPermutation(data, E_BIT_HOST, 48);
	uint64_t KxorER = ER ^ key;
	uint64_t S[8];
	uint64_t B[8];
	for (int i = 0; i < 8; i++)
	{
		B[i] = 0;
			
		for (int j = 1; j <= 6; j++)
		{
			SetNBit(&B[i], j, GetNBit(KxorER, i * 6 + j));
		}
		uint64_t firstLastBit = GetNBit(B[i], 1) << 1 | GetNBit(B[i], 6);
		uint64_t midBits = GetNBit(B[i],2) << 3 | GetNBit(B[i], 3) << 2 | GetNBit(B[i], 4) << 1 | GetNBit(B[i],5);
		S[i] = ALL_S_HOST[i][(int)firstLastBit * 16 + (int)midBits];
	}
	uint64_t result = 0;

	for (int i = 0; i < 8; i++)
	{
		result |= S[i] << 60 - 4 * i;

	}	
	return ApplyPermutation(result, P_HOST, 32);
}

#pragma endregion