#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdlib>
#include <iostream>
#include <chrono>
#include <random>
#include <ctime>
#include <cstdio>
#include "Constants.cuh"
#include "deviceAndHostFunctions.cuh"
#include "deviceFunctions.cuh"
#include "hostFunctions.cuh"

#pragma region CUDA_WRAPPERS

void cudaCheckErrors(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess) {
		std::cout << hipGetErrorString(cudaStatus) << std::endl;
		exit(1);
	}
}
#pragma endregion

//__device__ __host__ uint64_t GetNBit(uint64_t number, int bitNumber);
//__device__ __host__ void SetNBit(uint64_t* number, int bitNumber, uint64_t value);
//__device__ __host__ uint64_t ApplyPermutation(uint64_t number, int* Permutation_Table, int length);
//__device__ __host__ void SplitInHalf(uint64_t key, uint64_t* left, uint64_t* right, int keyLength);
//__device__ __host__ uint64_t CycleToLeft(uint64_t value, int shiftNumber, int valueLength);
//
//__host__ uint64_t EncryptData(uint64_t dataToEncrypt, uint64_t desKey);
//__host__  void GenerateSubKeys(uint64_t* subKeys, uint64_t desKey);
//__host__  void GenerateKn(uint64_t* subkeys, uint64_t* C, uint64_t* D);
//__host__ uint64_t Function(uint64_t data, uint64_t key);
//__host__ uint64_t Encode(uint64_t* subKeys, uint64_t dataToEncrypt);
//__host__ void Crack_Host(uint64_t* crackedKey, uint64_t dataToEncrypt, uint64_t encryptedMessage, uint64_t maxKeyVal, int keyLength);
//
//__device__ uint64_t EncryptData_Device(uint64_t dataToEncrypt, uint64_t desKey);
//__device__  void GenerateSubKeys_Device(uint64_t* subKeys, uint64_t desKey);
//__device__  void GenerateKn_Device(uint64_t* subkeys, uint64_t* C, uint64_t* D);
//__device__ uint64_t Function_Device(uint64_t data, uint64_t key);
//__device__ uint64_t Encode_Device(uint64_t* subKeys, uint64_t dataToEncrypt);

__host__ void Crack_Host(uint64_t* crackedKey, uint64_t dataToEncrypt, uint64_t encryptedMessage, uint64_t maxKeyVal, int keyLength);
__host__ void PrintUint(uint64_t v);
__global__ void Crack_Kernel(uint64_t data, uint64_t encodedData, uint64_t *crackedkey, bool *foundFlag, uint64_t maxKeyVal, int keyLength);
__host__ uint64_t GenerateDesKey(int keyLenght);


int main()
{

	cudaCheckErrors(hipSetDevice(0));
	std::cout << "Dlugosc klucza:" << std::endl;

	int keyLength;
	std::cin >> keyLength;
	uint64_t maxKeyVal = (uint64_t)1 << keyLength;
	uint64_t desKey = GenerateDesKey(keyLength);
	uint64_t dataToEncrypt = 0x0123456789ABCDEF;
	uint64_t encryptedMessage = EncryptData(dataToEncrypt, desKey);
	
	uint64_t* deviceKey = NULL, crackedKeyGPU;
	int cracked_val = 0;
	bool *wasCracked = NULL;
	cudaCheckErrors(hipMalloc((void**)&deviceKey, sizeof(uint64_t)));
	cudaCheckErrors(hipMalloc((void**)&wasCracked, sizeof(int)));
	cudaCheckErrors(hipMemcpy(wasCracked, &cracked_val, sizeof(int), hipMemcpyHostToDevice));

	std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
	Crack_Kernel << <4096, 1024 >> > (dataToEncrypt, encryptedMessage, deviceKey, wasCracked, maxKeyVal, keyLength);
	cudaCheckErrors(hipDeviceSynchronize());
	std::chrono::system_clock::time_point end = std::chrono::system_clock::now();

	auto gpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000000.0;

	cudaCheckErrors(hipMemcpy(&crackedKeyGPU, deviceKey, sizeof(uint64_t), hipMemcpyDeviceToHost));

	uint64_t encryptedDataWithKeyFromGPU = EncryptData(dataToEncrypt, crackedKeyGPU);
	if (encryptedDataWithKeyFromGPU == encryptedMessage)
	{
		std::cout << "GPU klucz znaleziony w: " << gpuExecutionTime << " sekund" << std::endl;
		std::cout << "Klucz znaleziony na GPU: " << (crackedKeyGPU >> (MAXL - keyLength));
		PrintUint(crackedKeyGPU);
		std::cout << "Oryginalny klucz: " <<(desKey >> (MAXL - keyLength));
		PrintUint(desKey);
	}
	else if (crackedKeyGPU == 0)
	{
		std::cout << "GPU nie znalazlo klucza." << std::endl << std::endl;
	}
	else
	{
		std::cout << "GPU klucz nie dziala." << std::endl;
	}
	std::cout << "================================================= " << std::endl << std::endl;


	start = std::chrono::system_clock::now();
	uint64_t crackedKeyCPU = -1;
	Crack_Host(&crackedKeyCPU, dataToEncrypt, encryptedMessage, maxKeyVal, keyLength);

	end = std::chrono::system_clock::now();

	auto cpuExecutionTime = (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000000.0;

	if (crackedKeyCPU != -1)
	{
		std::cout << "CPU klucz znaleziony w: " << cpuExecutionTime << " sekund" << std::endl;
		std::cout << "Klucz znaleziony na CPU: " << (crackedKeyCPU >> (MAXL - keyLength));
		PrintUint(crackedKeyCPU);
		std::cout << "Oryginalny klucz: " << (desKey >> (MAXL - keyLength));
		PrintUint(desKey);
	}
	else
	{
		std::cout << "CPU klucz nie dziala." << std::endl;
	}

	std::cout << "GPU znajduje klucz w " << gpuExecutionTime / cpuExecutionTime * 100 << " % czasu CPU." << std::endl;

	hipFree(deviceKey);
	hipFree(wasCracked);

	return 0;
}

__global__ void Crack_Kernel(uint64_t data, uint64_t encodedData, uint64_t *crackedkey, bool *foundFlag, uint64_t maxKeyVal, int keyLength)
{
	for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i <= maxKeyVal; i += blockDim.x * gridDim.x)
	{
		uint64_t keycandidate = i << (MAXL - keyLength);
		uint64_t currentValue = EncryptData_Device(data, keycandidate);
		if (currentValue == encodedData)
		{
			*crackedkey = keycandidate;
			*foundFlag = false;
			return;
		}
		if (*foundFlag == true)
		{
			return;
		}
	}
}

__host__ void Crack_Host(uint64_t* crackedKey, uint64_t dataToEncrypt, uint64_t encryptedMessage, uint64_t maxKeyVal, int keyLength)
{
	for (uint64_t i = 0; i < maxKeyVal; i++)
	{
		uint64_t keycandidate = i << (MAXL - keyLength);
		uint64_t currentValue = EncryptData(dataToEncrypt, keycandidate);
		if (currentValue == encryptedMessage)
		{
			*crackedKey = keycandidate;
			break;
		}
	}
}

__host__ void PrintUint(uint64_t v)
{
	std::cout << "\n";
	uint64_t j = 1;
	for (int i = 0; i < 64; i++)
	{
		std::cout << (v>>(63-i) &j);
		if ((i + 1) % 8 == 0)
			std::cout << " ";
	}
	std::cout << "\n";

}

//#pragma region DeviceAndHostFunctions
//
//
//__device__ __host__ uint64_t GetNBit(uint64_t number, int bitNumber)
//{
//	return ((uint64_t)1 & (number>>(MAXL-bitNumber)));
//}
//
//__device__ __host__ void SetNBit(uint64_t* number, int bitNumber, uint64_t value)
//{
//	(*number) = (*number) &  ~((uint64_t)1 << (MAXL - bitNumber));
//	(*number)= (*number) | (value << (MAXL - bitNumber));
//}
//
//__device__ __host__ uint64_t ApplyPermutation(uint64_t number, int* Permutation_Table, int length)
//{
//	uint64_t numberchanged = 0;
//	for (int i = 0; i < length; i++)
//	{
//		SetNBit(&numberchanged, i+1, GetNBit(number, Permutation_Table[i]));
//	}
//	return numberchanged;	
//}
//
//__device__ __host__ void SplitInHalf(uint64_t key, uint64_t* left, uint64_t* right, int keyLength)
//{
//	*right = *left = 0;
//	for (int i = 1; i <= keyLength / 2; i++)
//	{
//		SetNBit(right, i, GetNBit(key, keyLength / 2 + i));
//		SetNBit(left, i, GetNBit(key, i));
//	}
//}
//
//__device__ __host__ uint64_t CycleToLeft(uint64_t value, int shiftNumber, int valueLength) 
//{
//	for (int i = 0; i < shiftNumber; i++)
//	{
//		uint64_t bit = GetNBit(value, 1);
//		value <<= 1;
//		SetNBit(&value, valueLength, bit);
//	}
//	return value;
//}
//
//#pragma endregion

//#pragma region DeviceFunctions
//
//
//__device__  uint64_t EncryptData_Device(uint64_t dataToEncrypt, uint64_t desKey)
//{
//	uint64_t subKeys[16];
//
//	GenerateSubKeys_Device(subKeys, desKey);
//	uint64_t  encoded = Encode_Device(subKeys, dataToEncrypt);
//	return encoded;
//}
//
//__device__  void GenerateSubKeys_Device(uint64_t* subKeys, uint64_t desKey)
//{
//	uint64_t kplus = ApplyPermutation(desKey, PC_1, 56);
//	uint64_t C[17];
//	uint64_t D[17];
//
//	SplitInHalf(kplus, &C[0], &D[0], 56);
//
//	for (int i = 1; i <= 16; i++)
//	{
//		C[i] = CycleToLeft(C[i - 1], SHIFTS[i - 1], 28);
//		D[i] = CycleToLeft(D[i - 1], SHIFTS[i - 1], 28);
//	}
//
//	GenerateKn_Device(subKeys, C, D);
//}
//
//__device__  void GenerateKn_Device(uint64_t* subkeys, uint64_t* C, uint64_t* D)
//{
//	for (int i = 0; i < 16; i++)
//	{
//		subkeys[i] = C[i + 1];
//		subkeys[i] = subkeys[i] | (D[i + 1] >> 28);
//		subkeys[i] = ApplyPermutation(subkeys[i], PC_2, 48);
//	}
//
//}
//
//__device__  uint64_t Encode_Device(uint64_t* subKeys, uint64_t data)
//{
//	uint64_t data_ip = ApplyPermutation(data, IP, 64);
//
//	uint64_t L[17];
//	uint64_t R[17];
//
//	SplitInHalf(data_ip, &L[0], &R[0], 64);
//
//
//	for (int i = 1; i <= 16; i++)
//	{
//
//		L[i] = R[i - 1];
//		R[i] = L[i - 1] ^ Function_Device(R[i - 1], subKeys[i - 1]);
//	}
//	uint64_t RL = R[16] | (L[16] >> 32);
//	return ApplyPermutation(RL, IP_REV, 64);
//}
//
//__device__  uint64_t Function_Device(uint64_t data, uint64_t key)
//{
//	uint64_t ER = ApplyPermutation(data, E_BIT, 48);
//	uint64_t KxorER = ER ^ key;
//	uint64_t S[8];
//	uint64_t B[8];
//	for (int i = 0; i < 8; i++)
//	{
//		B[i] = 0;
//
//		for (int j = 1; j <= 6; j++)
//		{
//			SetNBit(&B[i], j, GetNBit(KxorER, i * 6 + j));
//		}
//		uint64_t firstLastBit = GetNBit(B[i], 1) << 1 | GetNBit(B[i], 6);
//		uint64_t midBits = GetNBit(B[i], 2) << 3 | GetNBit(B[i], 3) << 2 | GetNBit(B[i], 4) << 1 | GetNBit(B[i], 5);
//		S[i] = ALL_S[i][(int)firstLastBit * 16 + (int)midBits];
//	}
//	uint64_t result = 0;
//
//	for (int i = 0; i < 8; i++)
//	{
//		result |= S[i] << 60 - 4 * i;
//
//	}
//	return ApplyPermutation(result, P, 32);
//}
//
//
//#pragma endregion

//#pragma region HostFunctions
//
//__host__ uint64_t GenerateDesKey(int keyLenght)
//{
//	std::mt19937 mt(time(0));
//	std::uniform_int_distribution<int> randomV(0, 1);
//
//	uint64_t key = 0;
//	for (int i = 1; i <= keyLenght; i++)
//	{
//		SetNBit(&key,i, randomV(mt));
//	}
//	return key;
//}
//
//__host__  uint64_t EncryptData(uint64_t dataToEncrypt, uint64_t desKey)
//{
//	uint64_t subKeys[16];
//
//	GenerateSubKeys(subKeys, desKey);
//	return Encode(subKeys, dataToEncrypt);
//}
//
//__host__  void GenerateSubKeys(uint64_t* subKeys, uint64_t desKey)
//{
//	uint64_t kplus = ApplyPermutation(desKey, PC_1_HOST, 56);
//	uint64_t C[17];
//	uint64_t D[17];
//
//	SplitInHalf(kplus, &C[0], &D[0], 56);
//
//	for (int i = 1; i <= 16; i++)
//	{
//		C[i] = CycleToLeft(C[i - 1], SHIFTS_HOST[i-1], 28);
//		D[i] = CycleToLeft(D[i - 1], SHIFTS_HOST[i-1], 28);
//	}
//	
//	GenerateKn(subKeys,C, D);
//}
//
//__host__  void GenerateKn(uint64_t* subkeys, uint64_t* C, uint64_t* D)
//{
//	
//	for (int i = 0; i < 16; i++)
//	{
//		subkeys[i] = C[i + 1];
//		subkeys[i] = subkeys[i] | (D[i + 1] >> 28);
//		subkeys[i] = ApplyPermutation(subkeys[i], PC_2_HOST, 48);
//	}
//	
//}
//
//__host__  uint64_t Encode(uint64_t* subKeys, uint64_t data)
//{
//	uint64_t data_ip = ApplyPermutation(data, IP_HOST, 64);
//
//	uint64_t L[17];
//	uint64_t R[17];
//
//	SplitInHalf(data_ip, &L[0], &R[0], 64);
//
//
//	for (int i = 1; i <= 16; i++)
//	{
//
//		L[i] = R[i - 1];
//		R[i] = L[i - 1] ^ Function(R[i - 1], subKeys[i - 1]);
//	}
//	uint64_t RL = R[16] | (L[16] >> 32);
//	return ApplyPermutation(RL, IP_REV_HOST, 64);
//}
//
//__host__  uint64_t Function(uint64_t data, uint64_t key)
//{
//	uint64_t ER = ApplyPermutation(data, E_BIT_HOST, 48);
//	uint64_t KxorER = ER ^ key;
//	uint64_t S[8];
//	uint64_t B[8];
//	for (int i = 0; i < 8; i++)
//	{
//		B[i] = 0;
//			
//		for (int j = 1; j <= 6; j++)
//		{
//			SetNBit(&B[i], j, GetNBit(KxorER, i * 6 + j));
//		}
//		uint64_t firstLastBit = GetNBit(B[i], 1) << 1 | GetNBit(B[i], 6);
//		uint64_t midBits = GetNBit(B[i],2) << 3 | GetNBit(B[i], 3) << 2 | GetNBit(B[i], 4) << 1 | GetNBit(B[i],5);
//		S[i] = ALL_S_HOST[i][(int)firstLastBit * 16 + (int)midBits];
//	}
//	uint64_t result = 0;
//
//	for (int i = 0; i < 8; i++)
//	{
//		result |= S[i] << 60 - 4 * i;
//
//	}	
//	return ApplyPermutation(result, P_HOST, 32);
//}

#pragma endregion